#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iomanip> // Per una stampa più ordinata
#include <map>
#include <algorithm>
#include <set>
#include <unordered_set>
#include <chrono>
#include <thread>
#include <atomic>
#include <fstream>
#include <iostream>
#include "../utils/utils.h"
#include "../utils/matrixMath.h"

#define DEBUG 0

#define MAX_SHARED_LIST_PER_EDGE_COMBINED 16

#define CUDA_CHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}

#define DEBUG 0

using namespace std;

struct Edge{
    int v0;
    int v1;
};


//1 BLOCK FOR EACH EDGE
__global__ void EdgeIteratorAlgorithmKernel(
    int numEdges,
    const int* d_adjacencyList_rowPtr,
    const int* d_adjacencyList_colIdx,
    const Edge *d_edgeVector,
    const int* d_ranks,
    int* d_countTriangles
) {
    // Un blocco intero processa un solo arco. L'indice dell'arco è l'indice del blocco.
    int edge_idx = blockIdx.x;

    // Dichiarazione della memoria condivisa per le due liste di adiacenza.
    extern __shared__ int shared_lists[];

    if (edge_idx < numEdges) {
        Edge currentEdge = d_edgeVector[edge_idx];
        int v0 = currentEdge.v0;
        int v1 = currentEdge.v1;

        int rank_v0 = d_ranks[v0];
        int rank_v1 = d_ranks[v1];

        // Orienta l'arco dal nodo con rank più basso a quello con rank più alto
        if (rank_v0 > rank_v1) {
            int temp_v = v0; v0 = v1; v1 = temp_v;
            int temp_r = rank_v0; rank_v0 = rank_v1; rank_v1 = temp_r;
        }

        int v0_start = d_adjacencyList_rowPtr[v0];
        int v0_end = d_adjacencyList_rowPtr[v0 + 1];
        int v1_start = d_adjacencyList_rowPtr[v1];
        int v1_end = d_adjacencyList_rowPtr[v1 + 1];

        int len0 = v0_end - v0_start;
        int len1 = v1_end - v1_start;

        // Se le liste sono abbastanza piccole da entrare nella shared memory
        if (len0 > 0 && len1 > 0 && (len0 + len1) <= MAX_SHARED_LIST_PER_EDGE_COMBINED) {
            
            // **Caricamento Cooperativo**
            // I thread del blocco caricano la prima lista
            for (int i = threadIdx.x; i < len0; i += blockDim.x) {
                shared_lists[i] = d_adjacencyList_colIdx[v0_start + i];
            }
            // I thread del blocco caricano la seconda lista, posizionandola dopo la prima
            for (int i = threadIdx.x; i < len1; i += blockDim.x) {
                shared_lists[len0 + i] = d_adjacencyList_colIdx[v1_start + i];
            }
            
            // Sincronizza tutti i thread del blocco per assicurarsi che il caricamento sia completo
            __syncthreads();

            // **Intersezione in Shared Memory**
            // Un solo thread (il primo) esegue l'algoritmo di merge
            if (threadIdx.x == 0) {
                int p0 = 0;
                int p1 = 0;
                while (p0 < len0 && p1 < len1) {
                    int neighbor_v0 = shared_lists[p0];
                    // La seconda lista inizia dopo len0 elementi
                    int neighbor_v1 = shared_lists[len0 + p1]; 

                    if (neighbor_v0 == neighbor_v1) {
                        if (d_ranks[neighbor_v0] > rank_v1) {
                            atomicAdd(d_countTriangles, 1);
                        }
                        p0++;
                        p1++;
                    } else if (neighbor_v0 < neighbor_v1) {
                        p0++;
                    } else {
                        p1++;
                    }
                }
            }
        } else { // Fallback su memoria globale per liste troppo grandi
            // Un solo thread esegue il calcolo per evitare accessi multipli
            if (threadIdx.x == 0) {
                int p0 = v0_start;
                int p1 = v1_start;
                while (p0 < v0_end && p1 < v1_end) {
                    int neighbor_v0 = d_adjacencyList_colIdx[p0];
                    int neighbor_v1 = d_adjacencyList_colIdx[p1];

                    if (neighbor_v0 == neighbor_v1) {
                        if (d_ranks[neighbor_v0] > rank_v1) {
                            atomicAdd(d_countTriangles, 1);
                        }
                        p0++;
                        p1++;
                    } else if (neighbor_v0 < neighbor_v1) {
                        p0++;
                    } else {
                        p1++;
                    }
                }
            }
        }
    }
}


bool operator==(const Edge &e1, const Edge &e2) {
    return (e1.v0 == e2.v0 && e1.v1 == e2.v1) || (e1.v0 == e2.v1 && e1.v1 == e2.v0);
}

namespace std {
    template<>
    struct hash<Edge> {
        size_t operator()(const Edge& e) const {
            // Ordina i nodi per garantire che (u,v) e (v,u) abbiano lo stesso hash.
            int first = min(e.v0, e.v1);
            int second = max(e.v0, e.v1);

            size_t h1 = hash<int>{}(first);
            size_t h2 = hash<int>{}(second);
            
            return h1 ^ (h2 << 1); 
        }
    };
}


void createOrderedList(const map<int, vector<int>> &adjacencyVectors, vector<int> &orderedList){
    //create a map to store the degree of each node, then sort it
    map<int, int> nodeDegree;
    for (const auto &keyvaluepair: adjacencyVectors) {
        int node = keyvaluepair.first;
        int degree = keyvaluepair.second.size();
        nodeDegree[node] = degree;
    }
    //sort map based on degree
    vector<pair<int, int>> nodeDegreeSorted(nodeDegree.begin(), nodeDegree.end());
    sort(nodeDegreeSorted.begin(), nodeDegreeSorted.end(), [](const pair<int, int> &a, const pair<int, int> &b) {
        return a.second > b.second;
    });

    //just return the keys in the sorted order
    for (const auto &keyvaluepair : nodeDegreeSorted) {
        orderedList.emplace_back(keyvaluepair.first);
    }   

}


unordered_set<Edge> createEdgeSet(map<int, vector<int>> &adjacencyVectors) {
    unordered_set<Edge> edgeSet;

    for (const auto &keyvaluepair : adjacencyVectors) {
        int u = keyvaluepair.first;
        for (int v : keyvaluepair.second) {
            edgeSet.insert({u, v});
        }
    }

    return edgeSet;
}



int main(void){

    std::string input;
    while(true) {
        cout << "insert file name: ";
        std::getline(std::cin, input);
        input = "../graph_file/" + input;
        
        // check whether file can be opened
        std::ifstream file(input);
        
        if (file.is_open())
            break;
        cout << input << " doesn't exist!" << endl; 
    }


    // Crea la matrice di adiacenza NxN, inizializzata con tutti 0
    map<int, vector<int>> adjacencyVectors = populateAdjacencyVectors(input);
    vector<int> h_adjacencyList_rowPtr, h_adjacencyList_colIdx;
    int numNodes;

    convertToCRS(adjacencyVectors, h_adjacencyList_rowPtr, h_adjacencyList_colIdx, numNodes, true); //sort neighbors to speed up the algorithm and use merge-like approach in the kernel function


    vector<int> h_orderedList;
    createOrderedList(adjacencyVectors, h_orderedList);
    if (DEBUG) {

        std::cout << "Ordered list of nodes based on degree:\n";
        for (const auto &node : h_orderedList) {
            std::cout << node << " ";
        }
        std::cout << "\n";
    }


    //create edge set
    unordered_set<Edge> edgeSet = createEdgeSet(adjacencyVectors);

    //create vector of edges, since openmp works best with random access and vectors
    vector<Edge> h_edgeVector(edgeSet.begin(), edgeSet.end());
    int numEdges = edgeSet.size();


    //init ranks
    vector<int> h_ranks(numNodes + 1, 0); 
    for (int i = 0; i < h_orderedList.size(); ++i) {
        int nodeId = h_orderedList[i]; //node id
        int rank = i;                 //rank, based on the degree of the node, (i.e. the position in the ordered list, i.e. the number of neighbors it has)
        h_ranks[nodeId] = rank;
    }

    //ALLOC ON DEVICE
    int *d_adjacencyList_rowPtr, *d_adjacencyList_colIdx;
    Edge *d_edgeVector;
    int *d_ranks; 
    int* d_countTriangles;
    //mallocs
    CUDA_CHECK(hipMalloc(&d_adjacencyList_rowPtr, (numNodes +1) * sizeof(int) ));
    CUDA_CHECK(hipMalloc(&d_adjacencyList_colIdx, h_adjacencyList_colIdx.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_edgeVector, numEdges * sizeof(Edge)));
    CUDA_CHECK(hipMalloc(&d_ranks, numNodes * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_countTriangles, sizeof(int)));
    //copies
    CUDA_CHECK(hipMemcpy(d_edgeVector, h_edgeVector.data(), numEdges * sizeof(Edge), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_adjacencyList_rowPtr, h_adjacencyList_rowPtr.data(), (numNodes + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_adjacencyList_colIdx, h_adjacencyList_colIdx.data(), h_adjacencyList_colIdx.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ranks, h_ranks.data(), numNodes * sizeof(int), hipMemcpyHostToDevice));
    int h_countTriangles = 0;
    CUDA_CHECK(hipMemcpy(d_countTriangles, &h_countTriangles, sizeof(int), hipMemcpyHostToDevice));


    int blockSize = 24; //threads per block1
    int gridSize = numEdges;


    //start kernel function
    //number of bytes for shared memory
    size_t shmemBytes = blockSize * MAX_SHARED_LIST_PER_EDGE_COMBINED * sizeof(int);
    int maxSharedMemoryPerBlock;
    hipDeviceGetAttribute(&maxSharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    if (shmemBytes > maxSharedMemoryPerBlock) {
        std::cout << "Errore: la shared memory richiesta (" << shmemBytes << " bytes) supera il massimo del dispositivo (" << maxSharedMemoryPerBlock << " bytes)." << std::endl;
        shmemBytes = maxSharedMemoryPerBlock; // O gestisci l'errore
    }
    auto startTime = chrono::high_resolution_clock::now();
    EdgeIteratorAlgorithmKernel<<<gridSize, blockSize, shmemBytes>>>(
        numEdges,
        d_adjacencyList_rowPtr,
        d_adjacencyList_colIdx,
        d_edgeVector,
        d_ranks,
        d_countTriangles
    );

    CUDA_CHECK(hipDeviceSynchronize());

    //copy back result
    CUDA_CHECK(hipMemcpy(&h_countTriangles, d_countTriangles, sizeof(int), hipMemcpyDeviceToHost));

    auto endTime = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(endTime - startTime);
    std::cout << "-----------------------------------------------------------------" << std::endl;
    std::cout << "Time taken for edge iterator algorithm: " << duration.count() << " microseconds" << std::endl;
    std::cout << "Triangles found by edge iterator algorithm: " << h_countTriangles << std::endl;
    std::cout << "Total number of edges: " << numEdges << std::endl
              << "Total number of nodes: " << adjacencyVectors.size() << std::endl;


    //FREE DEVICE
    CUDA_CHECK(hipFree(d_adjacencyList_rowPtr));
    CUDA_CHECK(hipFree(d_adjacencyList_colIdx));
    CUDA_CHECK(hipFree(d_edgeVector));
    CUDA_CHECK(hipFree(d_ranks));
    CUDA_CHECK(hipFree(d_countTriangles));

    CUDA_CHECK(hipDeviceReset());


    return 0;
}