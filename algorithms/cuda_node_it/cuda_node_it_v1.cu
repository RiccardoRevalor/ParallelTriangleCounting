#include "hip/hip_runtime.h"
//NODE ITERATIR CUDA VERSION, WITH FOR LOOPS

#include <hip/hip_runtime.h>
#include <vector>
#include <iomanip> // Per una stampa più ordinata
#include <map>
#include <algorithm>
#include <set>
#include <unordered_set>
#include <chrono>
#include <thread>
#include <atomic>
#include <fstream>
#include <iostream>
#include "../../utils/utils.h"
#include "../../utils/matrixMath.h"

#define DEBUG 0
using namespace std;

#define CUDA_CHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}

__global__ void ForwardAlgorithmKernel(
    int numNodes,
    const int* d_adjacencyList_rowPtr,
    const int* d_adjacencyList_colIdx,
    const int* d_ranks,
    int* d_countTriangles
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numNodes) return;

    int s = tid;
    int rank_s = d_ranks[s];

    int s_start = d_adjacencyList_rowPtr[s];
    int s_end = d_adjacencyList_rowPtr[s + 1];

    // Iterate over neighbors of s
    for (int i = s_start; i < s_end; ++i) {
        int t = d_adjacencyList_colIdx[i];
        int rank_t_ = d_ranks[t];

        if (rank_s >= rank_t_) continue;

        int t_start = d_adjacencyList_rowPtr[t];
        int t_end = d_adjacencyList_rowPtr[t + 1];

        //for each neighbor t, iterate over its neighbors
        for (int j = t_start; j < t_end; ++j) {
            int v = d_adjacencyList_colIdx[j];

            if (rank_t_ >= d_ranks[v]) continue;

            for (int k = s_start; k < s_end; ++k) {
                if (d_adjacencyList_colIdx[k] == v) {
                    atomicAdd(d_countTriangles, 1);
                    break;
                }
            }
        }
    }
}

// Funzione per stampare la matrice di adiacenza
void printMatrix(const std::vector<std::vector<int>>& matrix) {
    int numVertices = matrix.size();
    
    // Stampa l'intestazione delle colonne
    std::cout << "   ";
    for (int i = 0; i < numVertices; ++i) {
        std::cout << std::setw(3) << i;
    }
    std::cout << "\n";
    std::cout << "---";
    for (int i = 0; i < numVertices; ++i) {
        std::cout << "---";
    }
    std::cout << "\n";

    // Stampa le righe della matrice
    for (int i = 0; i < numVertices; ++i) {
        std::cout << std::setw(2) << i << "|";
        for (int j = 0; j < numVertices; ++j) {
            std::cout << std::setw(3) << matrix[i][j];
        }
        std::cout << "\n";
    }
}


//Graphviz DOT format for printing the graph
void printDot(const std::vector<std::vector<int>>& matrix) {
    cout << "graph G {\n";

    int numVertices = matrix.size();
    for (int i = 0; i < numVertices; ++i) {
        for (int j = i + 1; j < numVertices; ++j) {
            if (matrix[i][j] == 1) {
                cout << "  " << i << " -- " << j << ";\n";
            }
        }
    }

    cout << "}\n";
}

void createOrderedList(const map<int, vector<int>> &adjacencyVectors, vector<int> &orderedList){
    //create a map to store the degree of each node, then sort it
    map<int, int> nodeDegree;
    for (const auto &keyvaluepair: adjacencyVectors) {
        int node = keyvaluepair.first;
        int degree = keyvaluepair.second.size();
        nodeDegree[node] = degree;
    }
    //sort map based on degree
    vector<pair<int, int>> nodeDegreeSorted(nodeDegree.begin(), nodeDegree.end());
    sort(nodeDegreeSorted.begin(), nodeDegreeSorted.end(), [](const pair<int, int> &a, const pair<int, int> &b) {
        return a.second > b.second;
    });

    //just return the keys in the sorted order
    for (const auto &keyvaluepair : nodeDegreeSorted) {
        orderedList.emplace_back(keyvaluepair.first);
    }   
}



int main() {

    std::string input;
    while(true) {
        cout << "insert file name: ";
        std::getline(std::cin, input);
        input = "../../graph_file/" + input;
        
        // check whether file can be opened
        std::ifstream file(input);
        
        if (file.is_open())
            break;
        cout << input << " doesn't exist!" << endl; 
    }


    // Crea la matrice di adiacenza NxN, inizializzata con tutti 0
    map<int, vector<int>> adjacencyVectors = populateAdjacencyVectors(input);
    vector<int> h_adjacencyList_rowPtr, h_adjacencyList_colIdx;
    int numNodes;

    convertToCRS(adjacencyVectors, h_adjacencyList_rowPtr, h_adjacencyList_colIdx, numNodes);


    vector<int> h_orderedList;
    createOrderedList(adjacencyVectors, h_orderedList);
    if (DEBUG) {

        std::cout << "Ordered list of nodes based on degree:\n";
        for (const auto &node : h_orderedList) {
            std::cout << node << " ";
        }
        std::cout << "\n";
    }

    //init ranks
    vector<int> h_ranks(numNodes + 1, 0); 
    for (int i = 0; i < h_orderedList.size(); ++i) {
        int nodeId = h_orderedList[i]; //node id
        int rank = i;                 //rank, based on the degree of the node, (i.e. the position in the ordered list, i.e. the number of neighbors it has)
        h_ranks[nodeId] = rank;
    }


    cout << "-----------------------------------------------------------------" << endl;
    //ALLOC ON DEVICE
    int *d_adjacencyList_rowPtr, *d_adjacencyList_colIdx;
    int *d_ranks; 
    int* d_countTriangles;
    //mallocs
    CUDA_CHECK(hipMalloc(&d_adjacencyList_rowPtr, (numNodes +1) * sizeof(int) ));
    CUDA_CHECK(hipMalloc(&d_adjacencyList_colIdx, h_adjacencyList_colIdx.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_ranks, numNodes * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_countTriangles, sizeof(int)));
    //copies
    CUDA_CHECK(hipMemcpy(d_adjacencyList_rowPtr, h_adjacencyList_rowPtr.data(), (numNodes + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_adjacencyList_colIdx, h_adjacencyList_colIdx.data(), h_adjacencyList_colIdx.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ranks, h_ranks.data(), numNodes * sizeof(int), hipMemcpyHostToDevice));
    int h_countTriangles = 0;
    CUDA_CHECK(hipMemcpy(d_countTriangles, &h_countTriangles, sizeof(int), hipMemcpyHostToDevice));

    int blockSize = 8; //threads per block
    int gridSize = (numNodes + blockSize - 1) / blockSize; //blocks in grid

    auto startTime = chrono::high_resolution_clock::now();

    ForwardAlgorithmKernel<<<gridSize, blockSize>>>(numNodes, d_adjacencyList_rowPtr, d_adjacencyList_colIdx, d_ranks, d_countTriangles);

    CUDA_CHECK(hipDeviceSynchronize());

    //copy back result
    CUDA_CHECK(hipMemcpy(&h_countTriangles, d_countTriangles, sizeof(int), hipMemcpyDeviceToHost));

    auto endTime = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(endTime - startTime);
    cout << "Time taken for forward algorithm: " << duration.count() << " microseconds" << endl;
    cout << "Triangles found by forward algorithm: " << h_countTriangles << endl;
    std::cout << "Total number of nodes: " << numNodes << std::endl;

    //FREE DEVICE
    CUDA_CHECK(hipFree(d_adjacencyList_rowPtr));
    CUDA_CHECK(hipFree(d_adjacencyList_colIdx));
    CUDA_CHECK(hipFree(d_ranks));
    CUDA_CHECK(hipFree(d_countTriangles));

    CUDA_CHECK(hipDeviceReset());

    return 0;
}