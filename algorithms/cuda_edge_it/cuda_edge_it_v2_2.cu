#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iomanip> // Per una stampa più ordinata
#include <map>
#include <algorithm>
#include <set>
#include <unordered_set>
#include <chrono>
#include <thread>
#include <atomic>
#include <fstream>
#include <iostream>
#include "../../utils/utils.h"
#include "../../utils/matrixMath.h"

#define DEBUG 0

#define CUDA_CHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}

#define DEBUG 0

using namespace std;

struct Edge{
    int v0;
    int v1;
};


//1 THREAD MANAGES JUST 1 EDGE
__global__ void EdgeIteratorAlgorithmKernel(
    int numEdges,
    const int* d_adjacencyList_rowPtr, // For CSR format
    const int* d_adjacencyList_colIdx, // For CSR format
    const Edge *d_edgeVector,
    const int* d_ranks,
    int* d_countTriangles,
    int MAX_SHARED_LIST_PER_EDGE_COMBINED = 32
) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numEdges){

        Edge currentEdge = d_edgeVector[idx];
        int v0 = currentEdge.v0;
        int v1 = currentEdge.v1;

        int rank_v0 = d_ranks[v0];
        int rank_v1 = d_ranks[v1];

        //swap in case v0 has higher rank than v1
        if (rank_v0 > rank_v1) {
            int temp = v0;
            v0 = v1;
            v1 = temp;
            int temp_rank = rank_v0;
            rank_v0 = rank_v1;
            rank_v1 = temp_rank;
        }

        //CSR standard
        int v0_start = d_adjacencyList_rowPtr[v0];
        int v0_end = d_adjacencyList_rowPtr[v0 + 1];

        int v1_start = d_adjacencyList_rowPtr[v1];
        int v1_end = d_adjacencyList_rowPtr[v1 + 1];

        //length of two lists
        int len0 = v0_end - v0_start;
        int len1 = v1_end - v1_start;
        //SHARED MEMORY IMPLEMENTATION
        if (len0 > 0 && len1 > 0 && (len0 + len1) <= MAX_SHARED_LIST_PER_EDGE_COMBINED) {
            // printf("Using shared memory for edge %d\n", idx); // Rimuovi in produzione per non rallentare
            
            extern __shared__ int shared_memory[]; // Shared memory dinamica per il blocco

            // Calcola l'offset per la sezione di shared memory di questo thread
            // Poiché un thread gestisce la propria edge, gli diamo uno spazio dedicato
            int thread_shmem_offset = threadIdx.x * MAX_SHARED_LIST_PER_EDGE_COMBINED;

            // Puntatori alle liste all'interno dello spazio shared memory di questo thread
            int *shared_list0 = shared_memory + thread_shmem_offset;
            int *shared_list1 = shared_memory + thread_shmem_offset + len0; //shared_list1 segue shared_list0 nello stesso spazio del thread

            // Load data into shared memory
            // Ogni thread carica la sua intera sezione
            for (int i = 0; i < len0; ++i) { // Ciclo for per ogni elemento della lista
                shared_list0[i] = d_adjacencyList_colIdx[v0_start + i];
            }
            for (int i = 0; i < len1; ++i) { // Ciclo for per ogni elemento della lista
                shared_list1[i] = d_adjacencyList_colIdx[v1_start + i];
            }
            __syncthreads(); // Sincronizza i thread del blocco dopo il caricamento

            // MERGE-LIKE ALGORITHM con puntatori RELATIVI per la shared memory
            int p0_sh = 0; // Puntatore per shared_list0, inizia da 0
            int p1_sh = 0; // Puntatore per shared_list1, inizia da 0

            while (p0_sh < len0 && p1_sh < len1){
                int neighbor_v0 = shared_list0[p0_sh];
                int neighbor_v1 = shared_list1[p1_sh];

                if (neighbor_v0 == neighbor_v1) {
                    if (d_ranks[neighbor_v0] > rank_v1) {
                        atomicAdd(d_countTriangles, 1);
                    }
                    p0_sh++;
                    p1_sh++;
                } else if (neighbor_v0 < neighbor_v1) {
                    p0_sh++;
                } else {
                    p1_sh++;
                }
            }
        } else {
            printf("Using global memory for edge %d\n", idx); // Rimuovi in produzione
            // Fallback to global memory access if shared memory is not suitable
            // MERGE-LIKE ALGORITHM con puntatori globali
            int p0 = v0_start; 
            int p1 = v1_start; 

            while (p0 < v0_end && p1 < v1_end){
                int neighbor_v0 = d_adjacencyList_colIdx[p0];
                int neighbor_v1 = d_adjacencyList_colIdx[p1];

                if (neighbor_v0 == neighbor_v1) {
                    if (d_ranks[neighbor_v0] > rank_v1) {
                        atomicAdd(d_countTriangles, 1);
                    }
                    p0++;
                    p1++;
                } else if (neighbor_v0 < neighbor_v1) {
                    p0++;
                } else {
                    p1++;
                }
            }
        } 
    }
}


bool operator==(const Edge &e1, const Edge &e2) {
    return (e1.v0 == e2.v0 && e1.v1 == e2.v1) || (e1.v0 == e2.v1 && e1.v1 == e2.v0);
}

namespace std {
    template<>
    struct hash<Edge> {
        size_t operator()(const Edge& e) const {
            // Ordina i nodi per garantire che (u,v) e (v,u) abbiano lo stesso hash.
            int first = min(e.v0, e.v1);
            int second = max(e.v0, e.v1);

            size_t h1 = hash<int>{}(first);
            size_t h2 = hash<int>{}(second);
            
            return h1 ^ (h2 << 1); 
        }
    };
}


void createOrderedList(const map<int, vector<int>> &adjacencyVectors, vector<int> &orderedList){
    //create a map to store the degree of each node, then sort it
    map<int, int> nodeDegree;
    for (const auto &keyvaluepair: adjacencyVectors) {
        int node = keyvaluepair.first;
        int degree = keyvaluepair.second.size();
        nodeDegree[node] = degree;
    }
    //sort map based on degree
    vector<pair<int, int>> nodeDegreeSorted(nodeDegree.begin(), nodeDegree.end());
    sort(nodeDegreeSorted.begin(), nodeDegreeSorted.end(), [](const pair<int, int> &a, const pair<int, int> &b) {
        return a.second > b.second;
    });

    //just return the keys in the sorted order
    for (const auto &keyvaluepair : nodeDegreeSorted) {
        orderedList.emplace_back(keyvaluepair.first);
    }   

}


unordered_set<Edge> createEdgeSet(map<int, vector<int>> &adjacencyVectors) {
    unordered_set<Edge> edgeSet;

    for (const auto &keyvaluepair : adjacencyVectors) {
        int u = keyvaluepair.first;
        for (int v : keyvaluepair.second) {
            edgeSet.insert({u, v});
        }
    }

    return edgeSet;
}



int main(int argc, char *argv[]) {

    if (argc != 5){
        cerr << "Usage: " << argv[0] << " <input_file> <BLOCK_SIZE> <MAX_SHARED_LIST_PER_EDGE_COMBINED> <GPU_MODEL>" << endl;
        return 1;
    }

    //if filename is "i" then ask for input
    std::string input;
    if (argv[1] == "i") {
        while (true) {
            std::cout << "insert file name: ";
            std::getline(std::cin, input);
            input = "../../graph_file/" + input;

            std::ifstream file(input);
            if (file.is_open())
                break;
            std::cout << input << " doesn't exist!" << std::endl;
        }
    } else {
        //extract file name from command line arguments
        input = "../../graph_file/" + std::string(argv[1]);
    }

    std::string gpuModel = argv[4];
    int blockSize = std::stoi(argv[2]);
    int MAX_SHARED_LIST_PER_EDGE_COMBINED = std::stoi(argv[3]);


    // Crea la matrice di adiacenza NxN, inizializzata con tutti 0
    map<int, vector<int>> adjacencyVectors = populateAdjacencyVectors(input);
    vector<int> h_adjacencyList_rowPtr, h_adjacencyList_colIdx;
    int numNodes;

    convertToCRS(adjacencyVectors, h_adjacencyList_rowPtr, h_adjacencyList_colIdx, numNodes, true); //sort neighbors to speed up the algorithm and use merge-like approach in the kernel function


    vector<int> h_orderedList;
    createOrderedList(adjacencyVectors, h_orderedList);
    if (DEBUG) {

        std::cout << "Ordered list of nodes based on degree:\n";
        for (const auto &node : h_orderedList) {
            std::cout << node << " ";
        }
        std::cout << "\n";
    }


    //create edge set
    unordered_set<Edge> edgeSet = createEdgeSet(adjacencyVectors);

    //create vector of edges, since openmp works best with random access and vectors
    vector<Edge> h_edgeVector(edgeSet.begin(), edgeSet.end());
    int numEdges = edgeSet.size();


    //init ranks
    vector<int> h_ranks(numNodes + 1, 0); 
    for (int i = 0; i < h_orderedList.size(); ++i) {
        int nodeId = h_orderedList[i]; //node id
        int rank = i;                 //rank, based on the degree of the node, (i.e. the position in the ordered list, i.e. the number of neighbors it has)
        h_ranks[nodeId] = rank;
    }

    //ALLOC ON DEVICE
    int *d_adjacencyList_rowPtr, *d_adjacencyList_colIdx;
    Edge *d_edgeVector;
    int *d_ranks; 
    int* d_countTriangles;
    //mallocs
    CUDA_CHECK(hipMalloc(&d_adjacencyList_rowPtr, (numNodes +1) * sizeof(int) ));
    CUDA_CHECK(hipMalloc(&d_adjacencyList_colIdx, h_adjacencyList_colIdx.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_edgeVector, numEdges * sizeof(Edge)));
    CUDA_CHECK(hipMalloc(&d_ranks, numNodes * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_countTriangles, sizeof(int)));
    //copies
    CUDA_CHECK(hipMemcpy(d_edgeVector, h_edgeVector.data(), numEdges * sizeof(Edge), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_adjacencyList_rowPtr, h_adjacencyList_rowPtr.data(), (numNodes + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_adjacencyList_colIdx, h_adjacencyList_colIdx.data(), h_adjacencyList_colIdx.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ranks, h_ranks.data(), numNodes * sizeof(int), hipMemcpyHostToDevice));
    int h_countTriangles = 0;
    CUDA_CHECK(hipMemcpy(d_countTriangles, &h_countTriangles, sizeof(int), hipMemcpyHostToDevice));


    int gridSize = (numEdges + blockSize - 1) / blockSize; //blocks in grid


    //start kernel function
    //number of bytes for shared memory
    size_t shmemBytes = blockSize * MAX_SHARED_LIST_PER_EDGE_COMBINED * sizeof(int);
    auto startTime = chrono::high_resolution_clock::now();
    EdgeIteratorAlgorithmKernel<<<gridSize, blockSize, shmemBytes>>>(
        numEdges,
        d_adjacencyList_rowPtr,
        d_adjacencyList_colIdx,
        d_edgeVector,
        d_ranks,
        d_countTriangles,
        MAX_SHARED_LIST_PER_EDGE_COMBINED
    );

    CUDA_CHECK(hipDeviceSynchronize());

    //copy back result
    CUDA_CHECK(hipMemcpy(&h_countTriangles, d_countTriangles, sizeof(int), hipMemcpyDeviceToHost));

    auto endTime = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(endTime - startTime);
    std::cout << "-----------------------------------------------------------------" << std::endl;
    std::cout << "Time taken for edge iterator algorithm: " << duration.count() << " microseconds" << std::endl;
    std::cout << "Triangles found by edge iterator algorithm: " << h_countTriangles << std::endl;
    std::cout << "Total number of edges: " << numEdges << std::endl
              << "Total number of nodes: " << adjacencyVectors.size() << std::endl;


    //FREE DEVICE
    CUDA_CHECK(hipFree(d_adjacencyList_rowPtr));
    CUDA_CHECK(hipFree(d_adjacencyList_colIdx));
    CUDA_CHECK(hipFree(d_edgeVector));
    CUDA_CHECK(hipFree(d_ranks));
    CUDA_CHECK(hipFree(d_countTriangles));

    CUDA_CHECK(hipDeviceReset());



    // create cross validation output file
    std::ofstream crossValidationFile;
    // Corrected string concatenation for filename

    //REMOVE .g extension from input file name
    size_t pos = input.find_last_of(".");
    if (pos != std::string::npos) {
        input = input.substr(0, pos);
    }
    //take just the file name without path
    pos = input.find_last_of("/");
    if (pos != std::string::npos) {
        input = input.substr(pos + 1);
    }
    string outputFileName("../../cross_validation_output/cuda_edge_it_v2_2/" + input + "_" + gpuModel + ".csv");
    cout << "Output file name: " << outputFileName << endl;

    crossValidationFile.open(outputFileName, std::ios::app);
    if (!crossValidationFile.is_open()) { // Use is_open() for robust check
        std::cerr << "Error opening cross validation output file!" << std::endl;
        return -1;
    }

    // write parameters and final time to the file, CSV format
    // put header if file is empty
    // Check if the file is empty by seeking to end and checking position
    crossValidationFile.seekp(0, std::ios::end); // Move to end
    if (crossValidationFile.tellp() == 0) { // Check position
        crossValidationFile << "BLOCK_SIZE,MAX_SHARED_LIST_PER_EDGE_COMBINED,GPU_MODEL,TOTAL_DURATION_US,TRIANGLES\n";
    }
    // Changed `duration` to `duration_mm` and added `duration_trace`
    crossValidationFile << blockSize << ","
                      << MAX_SHARED_LIST_PER_EDGE_COMBINED << ","
                      << gpuModel << ","
                      << duration.count() << ","
                      << countTriangles << "\n";

    crossValidationFile.close();


    return 0;
}